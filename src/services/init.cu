#include "hip/hip_runtime.h"
#include "init.h"
#include <blitz/array.h>
#include <limits>
#include "../constants.h"
#include "../data/tipsy.h"

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceInit::input>()  || std::is_trivial<ServiceInit::input>());
static_assert(std::is_void<ServiceInit::output>() || std::is_trivial<ServiceInit::output>());

static unsigned long x=123456789, y=362436069, z=521288629;

float xorshf96(void) {          //period 2^96-1
    unsigned long t;
    x ^= x << 16;
    x ^= x >> 5;
    x ^= x << 1;

    t = x;
    x = y;
    y = z;
    z = t ^ x ^ y;

    return (float) z / std::numeric_limits<unsigned long>::max() - 0.5;
}

int ServiceInit::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {

    printf("ServiceInit invoked on thread %d\n",pst->idSelf);

    auto lcl = pst->lcl;
    ServiceInit::input in = *static_cast<input *>(vin);

    // Init positions
    blitz::GeneralArrayStorage<2> storage;
    storage.ordering() = 0,1;
    storage.base() = 0, 0;
    storage.ascendingFlag() = true, true;
    // x, y, z, cellId, tmp
    int k = 3;
    auto particles = blitz::Array<float, 2>(in.nParticles, k, storage);

    float * particlesAxisData = (float *)calloc(N, sizeof(float ));
    CUDA_CHECK(hipHostMalloc, ((void**)&particlesAxisData, N * sizeof (float )));

    auto particlesAxis = blitz::Array<float, 1>(
            particlesAxisData,
            in.nParticles,
            blitz::deleteDataWhenDone);
    auto cellToRangeMap = blitz::Array<uint, 2>(MAX_CELLS, 2);
    float * d_particles;

    //auto tipsy = TipsyIO("../data/tipsy/b0-final.std");
    //tipsy.load()

    srand(pst->idSelf);
    int c = 0;
    for (int i = 0; i < in.nParticles; i++) {
        for (int d = 0; d < 3; d++) {
            particles(i,d) = xorshf96();
            if (particles(i,d) < 0.0) c++;
        }
    }

    printf("ServiceInit generated random numbers %d\n",pst->idSelf);

    cellToRangeMap(0, 0) = 0;
    cellToRangeMap(0, 1) = in.nParticles;

    const int nBlocks = (int) ceil((float) in.nParticles / (N_THREADS * ELEMENTS_PER_THREAD)) + MAX_CELLS;
    unsigned int* h_results = (unsigned int*)malloc(nBlocks * sizeof(unsigned int));
    CUDA_CHECK(hipHostMalloc, ((void**)&h_results, nBlocks * sizeof (unsigned int)));

    lcl->h_results = h_results;
    lcl->particles.reference(particles);
    lcl->particlesAxis.reference(particlesAxis);
    lcl->d_particles = d_particles;
    lcl->cellToRangeMap.reference(cellToRangeMap);

    CUDA_CHECK(hipMalloc,(&lcl->d_particles, sizeof (float) * in.nParticles));

    CUDA_CHECK(hipMalloc, (&lcl->d_results, sizeof (unsigned int) * nBlocks));

    auto streams = blitz::Array<hipStream_t , 1>(N_STREAMS);

    for (int i = 0; i < N_STREAMS; i++) {
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate, (&stream));
        streams(i) = stream;
    }

    lcl->streams.reference(streams);

    printf("ServiceInit finished on thread %d\n",pst->idSelf);

    return 0;
}

int ServiceInit::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {
    return 0;
}
