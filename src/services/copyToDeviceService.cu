#include "hip/hip_runtime.h"
#include "copyToDeviceService.h"
#include <blitz/array.h>
#include <vector>

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceCopyToDevice::input>()  || std::is_trivial<ServiceCopyToDevice::input>());
static_assert(std::is_void<ServiceCopyToDevice::output>() || std::is_trivial<ServiceCopyToDevice::output>());

int ServiceCopyToDevice::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local data
    //
    auto lcl = pst->lcl;
    auto in  = static_cast<input *>(vin);
    auto out = static_cast<output *>(vout);
    auto nCells = nIn / sizeof(input);
    assert(nOut / sizeof(output) >= nCells);
    //printf("ServiceCopyToDevice invoked on thread %d\n",pst->idSelf);

    const int nThreads = 512;
    // Can increase speed by another factor of around two
    const int elementsPerThread = 16;

    auto cell = static_cast<Cell>(*(in + cellPtrOffset));

    int nParticles = lcl->particles.rows();

    const int nBlocks = ceil(nParticles / (nThreads * elementsPerThread) / 2 );

    //hipStreamSynchronize(lcl->streams(streamId));
    int * d_counts;
    pst->lcl->d_counts(cellPtrOffset) = d_counts;

    blitz::Array<float,1> particles = pst->lcl->particles(blitz::Range(beginInd, endInd), 0);

    hipMalloc(&d_particles, sizeof (float) * nParticles);
    hipMalloc(&lcl->d_counts(cellPtrOffset), sizeof (int) * nBlocks);
    hipMemcpyAsync(
            d_particles,
            particles.data(),
            endInd - beginInd,
            hipMemcpyHostToDevice,
            pst->lcl->streams(streamId)
    );


    return nCells * sizeof(output);
}

int ServiceCopyToDevice::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {

    return 0;
}
