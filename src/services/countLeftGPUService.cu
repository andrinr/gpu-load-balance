#include "hip/hip_runtime.h"
#include "countLeftGPUService.h"
#include <blitz/array.h>
#include <array>

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceCountLeftGPU::input>()  || std::is_trivial<ServiceCountLeftGPU::input>());
static_assert(std::is_void<ServiceCountLeftGPU::output>() || std::is_trivial<ServiceCountLeftGPU::output>());

template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce(float *g_idata, uint *g_odata, float cut, int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    // todo: ask doug
    while (i < n) {
        sdata[tid] += (g_idata[i] < cut);

        if (i + blockSize < n) {
            sdata[tid] += (g_idata[i+blockSize] < cut);
        }
        i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        } __syncthreads();
    }
    if (tid < 32) {
        warpReduce<blockSize>(sdata, tid);
    }
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }

}

int ServiceCountLeftGPU::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local d
    // ata
    auto lcl = pst->lcl;
    auto in  = static_cast<input *>(vin);
    auto out = static_cast<output *>(vout);
    const int nCells = nIn / sizeof(input);
    assert(nOut / sizeof(output) >= nCells);

    //int bytes = nCounts * sizeof (uint);
    // https://developer.nvidia.com/blog/how-overlap-data-transfers-cuda-cc/
    int blockOffset = 0;
    std::array<int, MAX_CELLS> offsets;
    offsets[0] = 0;

    for (int cellPtrOffset = 0; cellPtrOffset < nCells; ++cellPtrOffset) {
        out[cellPtrOffset] = 0;
    }

    for (int cellPtrOffset = 0; cellPtrOffset < nCells; ++cellPtrOffset) {
        auto cell = static_cast<Cell>(*(in + cellPtrOffset));

        if (cell.foundCut) {
            offsets[cellPtrOffset+1] = blockOffset;
            continue;
        }
        int beginInd = pst->lcl->cellToRangeMap(cell.id, 0);
        int endInd =  pst->lcl->cellToRangeMap(cell.id, 1);
        int n = endInd - beginInd;
        float cut = cell.getCut();

        if (n > 1 << 12) {
            const int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0 * ELEMENTS_PER_THREAD));

            reduce<N_THREADS>
            <<<
                nBlocks,
                N_THREADS,
                N_THREADS * sizeof (uint),
                lcl->stream
            >>>
                (lcl->d_particles + beginInd,
                 lcl->d_counts + blockOffset,
                 cut,
                 n);

            blockOffset += nBlocks;
        }
        else {
            blitz::Array<float,1> particles =
                    pst->lcl->particles(blitz::Range(beginInd, endInd), 0);

            float * startPtr = particles.data();
            float * endPtr = startPtr + (endInd - beginInd);

            for(auto p= startPtr; p<endPtr; ++p)
            {
                out[cellPtrOffset] += *p < cut;
            }
        }

        offsets[cellPtrOffset+1] = blockOffset;
    }

    CUDA_CHECK(hipMemcpyAsync,(
            lcl->h_counts,
            lcl->d_counts,
            sizeof (uint) * blockOffset,
            hipMemcpyDeviceToHost,
            lcl->stream));

    CUDA_CHECK(hipStreamSynchronize,(lcl->stream));

    for (int cellPtrOffset = 0; cellPtrOffset < nCells; ++cellPtrOffset) {
        int begin = offsets[cellPtrOffset];
        int end = offsets[cellPtrOffset + 1];

        for (int i = begin; i < end; ++i) {
            out[cellPtrOffset] += lcl->h_counts[i];
        }
    }

    return nCells * sizeof(output);
}

int ServiceCountLeftGPU::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {
    auto out  = static_cast<output *>(vout);
    auto out2 = static_cast<output *>(vout2);
    int nCounts = nIn / sizeof(input);
    assert(nOut1 >= nCounts*sizeof(output));
    assert(nOut2 >= nCounts*sizeof(output));
    for(auto i=0; i<nCounts; ++i)
	    out[i] += out2[i];
    return nCounts * sizeof(output);
}
