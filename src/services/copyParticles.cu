#include "copyParticles.h"
#include <blitz/array.h>
#include <vector>
#include "../constants.h"
// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceCopyParticles::input>()  || std::is_trivial<ServiceCopyParticles::input>());
static_assert(std::is_void<ServiceCopyParticles::output>() || std::is_trivial<ServiceCopyParticles::output>());

int ServiceCopyParticles::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local data
    auto lcl = pst->lcl;
    ServiceCopyParticles::input in = *static_cast<input *>(vin);

    int nParticles = lcl->particles.rows();
    // We only need the first nParticles, since axis 0 is axis where cuts need to be found

    printf("nParticles: %d\n", nParticles);
    if (in.params.GPU_COUNT and not in.params.GPU_PARTITION) {
        hipMemcpyAsync(
                lcl->d_particlesT,
                lcl->particlesT.data(),
                sizeof (float) * nParticles,
                hipMemcpyHostToDevice,
                pst->lcl->streams(0)
        );
    }

    if (in.params.GPU_PARTITION) {
        blitz::Array<float, 1> x = lcl->particles(blitz::Range::all(), 0);
        blitz::Array<float, 1> y = lcl->particles(blitz::Range::all(), 1);
        blitz::Array<float, 1> z = lcl->particles(blitz::Range::all(), 2);

        CUDA_CHECK(hipMemcpyAsync,(
                lcl->d_particlesX,
                x.data(),
                sizeof (float) * nParticles,
                hipMemcpyHostToDevice,
                pst->lcl->streams(0)
        ));

        CUDA_CHECK(hipMemcpyAsync,(
                lcl->d_particlesY,
                y.data(),
                sizeof (float) * nParticles,
                hipMemcpyHostToDevice,
                pst->lcl->streams(0)
        ));

        CUDA_CHECK(hipMemcpyAsync,(
                lcl->d_particlesZ,
                z.data(),
                sizeof (float) * nParticles,
                hipMemcpyHostToDevice,
                pst->lcl->streams(0)
        ));
    }

    return sizeof(output);
}

int ServiceCopyParticles::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {

    return 0;
}
