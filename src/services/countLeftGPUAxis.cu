#include "hip/hip_runtime.h"
#include "countLeftGPUAxis.h"
#include <blitz/array.h>
#include <array>

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceCountLeftGPUAxis::input>()  || std::is_trivial<ServiceCountLeftGPUAxis::input>());
static_assert(std::is_void<ServiceCountLeftGPUAxis::output>() || std::is_trivial<ServiceCountLeftGPUAxis::output>());

template <unsigned int blockSize>
extern __device__ void warpReduce(volatile unsigned int *s_data, unsigned int tid) {
    if (blockSize >= 64) s_data[tid] += s_data[tid + 32];
    if (blockSize >= 32) s_data[tid] += s_data[tid + 16];
    if (blockSize >= 16) s_data[tid] += s_data[tid + 8];
    if (blockSize >= 8) s_data[tid] += s_data[tid + 4];
    if (blockSize >= 4) s_data[tid] += s_data[tid + 2];
    if (blockSize >= 2) s_data[tid] += s_data[tid + 1];
}

template <unsigned int blockSize>
extern __global__ void reduce(
        float * g_idata,
        unsigned int * g_begins,
        unsigned int * g_ends,
        float * g_cuts,
        unsigned int * g_odata) {

    __shared__ unsigned int s_data[blockSize];

    unsigned int tid = threadIdx.x;
    const unsigned int begin = g_begins[blockIdx.x];
    const unsigned int end = g_ends[blockIdx.x];
    const float cut = g_cuts[blockIdx.x];

    unsigned int i = begin + tid;
    s_data[tid] = 0;

    // unaligned coalesced g memory access
    while (i < end) {
        s_data[tid] += (g_idata[i] <= cut);
        i += blockSize;
    }
    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            s_data[tid] += s_data[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            s_data[tid] += s_data[tid + 128];
        } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            s_data[tid] += s_data[tid + 64];
        } __syncthreads();
    }
    if (tid < 32) {
        warpReduce<blockSize>(s_data, tid);
    }
    if (tid == 0) {
        g_odata[blockIdx.x] = s_data[0];
    }
}

int ServiceCountLeftGPUAxis::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local d
    // ata
    auto lcl = pst->lcl;
    auto in  = static_cast<input *>(vin);
    auto out = static_cast<output *>(vout);
    unsigned int nCells = nIn / sizeof(input);
    assert(nOut / sizeof(output) >= nCells);

    std::vector<unsigned  int> cellIndices ;

    int nBlocks = 0;
    int blockPtr = 0;
    for (int cellPtrOffset = 0; cellPtrOffset < nCells; ++cellPtrOffset) {
        auto cell = static_cast<Cell>(*(in + cellPtrOffset));
        unsigned int beginInd = pst->lcl->cellToRangeMap(cell.id, 0);
        unsigned int endInd =  pst->lcl->cellToRangeMap(cell.id, 1);
        unsigned int n = endInd - beginInd;

        unsigned int nBlocksPerCell = (int) floor((float) n / (N_THREADS * ELEMENTS_PER_THREAD));

        int begin = beginInd;
        for (int i = 0; i < nBlocksPerCell; ++i) {
            lcl->h_cuts[blockPtr] = cell.getCut();
            cellIndices.push_back(cellPtrOffset);
            blockPtr++;
        }
        nBlocks += nBlocksPerCell;

        out[cellPtrOffset] = 0;
    }

    CUDA_CHECK(hipMemcpyAsync,(
            lcl->d_cuts,
            lcl->h_cuts,
            sizeof (float) * nBlocks,
            hipMemcpyHostToDevice,
            lcl->streams(0)));

    //CUDA_CHECK(hipMemset, (lcl->d_index, 0, sizeof(unsigned int)));

    // Execute the kernel
    reduce<N_THREADS><<<
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (unsigned int),
            lcl->streams(0)
            >>>(
                lcl->d_particlesT,
                lcl->d_begins,
                lcl->d_ends,
                lcl->d_cuts,
                lcl->d_results
            );

    //
    CUDA_CHECK(hipMemcpyAsync,(
            lcl->h_results,
            lcl->d_results,
            sizeof (unsigned int) * nBlocks,
            hipMemcpyDeviceToHost,
            lcl->streams(0)
    ));

    CUDA_CHECK(hipStreamSynchronize,(lcl->streams(0)));

    for (int i = 0; i < nBlocks; ++i) {
        out[cellIndices[i]] += lcl->h_results[i];
    }

    return nCells * sizeof(output);
}

int ServiceCountLeftGPUAxis::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {
    auto out  = static_cast<output *>(vout);
    auto out2 = static_cast<output *>(vout2);
    int nCounts = nIn / sizeof(input);
    assert(nOut1 >= nCounts*sizeof(output));
    assert(nOut2 >= nCounts*sizeof(output));
    for(auto i=0; i<nCounts; ++i)
	    out[i] += out2[i];
    return nCounts * sizeof(output);
}
