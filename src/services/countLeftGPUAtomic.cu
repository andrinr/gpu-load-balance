#include "hip/hip_runtime.h"
#include "countLeftGPUAtomic.h"
#include <blitz/array.h>
#include <array>
#include "../utils/condReduce.cuh"

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceCountLeftGPUAtomic::input>()  || std::is_trivial<ServiceCountLeftGPUAtomic::input>());
static_assert(std::is_void<ServiceCountLeftGPUAtomic::output>() || std::is_trivial<ServiceCountLeftGPUAtomic::output>());

template <unsigned int blockSize>
extern __device__ void warpReduce(volatile unsigned int *s_data, unsigned int tid) {
    if (blockSize >= 64) s_data[tid] += s_data[tid + 32];
    if (blockSize >= 32) s_data[tid] += s_data[tid + 16];
    if (blockSize >= 16) s_data[tid] += s_data[tid + 8];
    if (blockSize >= 8) s_data[tid] += s_data[tid + 4];
    if (blockSize >= 4) s_data[tid] += s_data[tid + 2];
    if (blockSize >= 2) s_data[tid] += s_data[tid + 1];
}

template <unsigned int blockSize>
extern __global__ void reduce(
        float * g_idata,
        unsigned int * g_begins,
        unsigned int * g_ends,
        float * g_cuts,
        unsigned int * a_index,
        unsigned int * g_odata) {

    __shared__ unsigned int s_data[blockSize];
    __shared__ unsigned int s_index;

    unsigned int tid = threadIdx.x;

    if (tid == 0) {
        s_index = atomicAdd(a_index, 1);
    }
    __syncthreads();

    const unsigned int begin = g_begins[s_index];
    const unsigned int end = g_ends[s_index];
    const float cut = g_cuts[s_index];

    unsigned int i = begin + tid;
    //const unsigned int gridSize = blockSize*gridDim.x;
    s_data[tid] = 0;

    while (i < end) {
        s_data[tid] += (g_idata[i] <= cut);
        i += blockSize;
    }
    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            s_data[tid] += s_data[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            s_data[tid] += s_data[tid + 128];
        } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            s_data[tid] += s_data[tid + 64];
        } __syncthreads();
    }
    if (tid < 32) {
        warpReduce<blockSize>(s_data, tid);
    }
    if (tid == 0) {
        g_odata[s_index] = s_data[0];
    }
}

int ServiceCountLeftGPUAtomic::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local d
    // ata
    auto lcl = pst->lcl;
    auto in  = static_cast<input *>(vin);
    auto out = static_cast<output *>(vout);
    unsigned int nCells = nIn / sizeof(input);
    assert(nOut / sizeof(output) >= nCells);

    std::vector<unsigned  int> cellIndices ;

    int nBlocks = 0;
    int blockPtr = 0;
    for (int cellPtrOffset = 0; cellPtrOffset < nCells; ++cellPtrOffset) {
        auto cell = static_cast<Cell>(*(in + cellPtrOffset));
        unsigned int beginInd = pst->lcl->cellToRangeMap(cell.id, 0);
        unsigned int endInd =  pst->lcl->cellToRangeMap(cell.id, 1);
        unsigned int n = endInd - beginInd;

        unsigned int nBlocksPerCell = (int) ceil((float) n / (N_THREADS * ELEMENTS_PER_THREAD));

        int begin = beginInd;
        for (int i = 0; i < nBlocksPerCell; ++i) {
            lcl->h_cuts[blockPtr] = cell.getCut();
            lcl->h_begins[blockPtr] = begin;
            begin += N_THREADS * ELEMENTS_PER_THREAD;
            lcl->h_ends[blockPtr] = min(begin, endInd);
            cellIndices.push_back(cellPtrOffset);
            blockPtr++;
        }
        nBlocks += nBlocksPerCell;

        out[cellPtrOffset] = 0;
    }

    //printf("nBlocks: %d\n", nBlocks);
    CUDA_CHECK(hipMemcpyAsync,(
            lcl->d_begins,
            lcl->h_begins,
            sizeof (unsigned int) * nBlocks,
            hipMemcpyHostToDevice,
            lcl->streams(0)
    ));

    CUDA_CHECK(hipMemcpyAsync,(
            lcl->d_ends,
            lcl->h_ends,
            sizeof (unsigned int) * nBlocks,
            hipMemcpyHostToDevice,
            lcl->streams(0)));

    CUDA_CHECK(hipMemcpyAsync,(
            lcl->d_cuts,
            lcl->h_cuts,
            sizeof (float) * nBlocks,
            hipMemcpyHostToDevice,
            lcl->streams(0)));

    CUDA_CHECK(hipMemset, (lcl->d_index, 0, sizeof(unsigned int)));

    // Execute the kernel
    reduce<N_THREADS><<<
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (unsigned int),
            lcl->streams(0)
            >>>(
                lcl->d_particlesT,
                lcl->d_begins,
                lcl->d_ends,
                lcl->d_cuts,
                lcl->d_index,
                lcl->d_results
            );

    //
    CUDA_CHECK(hipMemcpyAsync,(
            lcl->h_results,
            lcl->d_results,
            sizeof (unsigned int) * nBlocks,
            hipMemcpyDeviceToHost,
            lcl->streams(0)));

    CUDA_CHECK(hipStreamSynchronize,(lcl->streams(0)));

    for (int i = 0; i < nBlocks; ++i) {
        out[cellIndices[i]] += lcl->h_results[i];
    }

    return nCells * sizeof(output);
}

int ServiceCountLeftGPUAtomic::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {
    auto out  = static_cast<output *>(vout);
    auto out2 = static_cast<output *>(vout2);
    int nCounts = nIn / sizeof(input);
    assert(nOut1 >= nCounts*sizeof(output));
    assert(nOut2 >= nCounts*sizeof(output));
    for(auto i=0; i<nCounts; ++i)
	    out[i] += out2[i];
    return nCounts * sizeof(output);
}
