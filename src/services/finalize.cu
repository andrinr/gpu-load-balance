#include "finalize.h"
#include <blitz/array.h>
#include <vector>

// Make sure that the communication structure is "trivial" so that it
// can be moved around with "memcpy" which is required for MDL.
static_assert(std::is_void<ServiceFinalize::input>()  || std::is_trivial<ServiceFinalize::input>());
static_assert(std::is_void<ServiceFinalize::output>() || std::is_trivial<ServiceFinalize::output>());

int ServiceFinalize::Service(PST pst,void *vin,int nIn,void *vout, int nOut) {
    // store streams / initialize in local data
    auto lcl = pst->lcl;
    ServiceFinalize::input in = *static_cast<input *>(vin);

    if (in.params.GPU_COUNT) {
        hipHostFree(lcl->particlesT.data());

        hipFree(lcl->d_particlesT);
        hipFree(lcl->d_results);
        hipHostFree(lcl->h_results);
    }

    if (in.params.GPU_COUNT_ATOMIC) {
        hipHostFree(lcl->h_cuts);
        hipHostFree(lcl->h_begins);
        hipHostFree(lcl->h_ends);
        hipFree(lcl->d_cuts);
        hipFree(lcl->d_begins);
        hipFree(lcl->d_ends);
        hipFree(lcl->d_index);
    }

    if (in.params.GPU_PARTITION) {
        hipFree(lcl->d_particlesX);
        hipFree(lcl->d_particlesY);
        hipFree(lcl->d_particlesZ);
        hipFree(lcl->d_offsetLeq);
        hipFree(lcl->d_offsetG);
    }


    return sizeof(output);
}

int ServiceFinalize::Combine(void *vout,void *vout2,int nIn,int nOut1,int nOut2) {

    return 0;
}
