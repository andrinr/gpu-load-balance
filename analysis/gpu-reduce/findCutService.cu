#include "hip/hip_runtime.h"
//
// Created by andrin on 15/05/22.
//
#include <blitz/array.h>

template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

// todo: Rewrite this to work with cells
template <unsigned int blockSize>
__global__ void reduce(float *g_idata, int *g_odata, float cut, int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        } __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        } __syncthreads();
    }
    if (tid < 32) {
        warpReduce<blockSize>(sdata, tid);
    }
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

int main(int argc, char** argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 1 << 27;
    int nd = 3;

    float * pos = (float*)calloc(n, sizeof(float));
    for (int i = 0; i < n; i++) {
        pos[i] = (float)(rand())/(float)(RAND_MAX);
    }

    int testSum = 0;
    for (int i = 0; i < 10000; i++) {
        testSum += pos[i] < 0.5;
    }
    std::cout << testSum << "\n";

    const int nThreads = 256;
    // Can increase speed by another factor of around two
    int elementsPerThread = 16;
    int nBlocks = ceil(n / nThreads / 2 / elementsPerThread);

    float * d_particles;
    int * d_sums;
    int * h_sums = (int*)calloc(n, sizeof(int));
    hipMalloc(&d_particles, sizeof (float) * n);
    hipMemcpy(d_particles, pos, sizeof (float ) * n, hipMemcpyHostToDevice);

    hipMalloc(&d_sums, sizeof (int) * n);

    // Number of threads per block is limited

    // Need for cut service becomes clear here!
    float cut = 0.5;

    hipEventRecord(start);
    reduce<nThreads><<<nBlocks, nThreads, nThreads * sizeof (int) >>>(d_particles, d_sums, cut, n);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    hipMemcpy(h_sums, d_sums, sizeof (int ) * nBlocks, hipMemcpyDeviceToHost);

    int sum = 0;

    for (int i = 0; i < nBlocks; ++i) {
        sum += h_sums[i];
    }

    std::cout << sum << " " << n << "\n";

    hipFree(d_particles);
    hipFree(d_sums);

    free(h_sums);
    free(pos);

    hipDeviceReset();
}
