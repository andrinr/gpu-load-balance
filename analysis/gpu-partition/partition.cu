#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include "../../src/utils/condReduce.cuh"
#include "../../src/constants.h"
#include <thrust/partition.h>
// https://www.cse.chalmers.se/~tsigas/papers/GPU-Quicksort-jea.pdf
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/2_Concepts_and_Techniques/scan
// https://onlinelibrary.wiley.com/doi/epdf/10.1002/cpe.3611

// 2 data elements per thread
// Code taken from: https://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
__device__ void scan(volatile unsigned int * s_idata, unsigned int thid, unsigned int n) {
    unsigned int offset = 1;
    for (unsigned int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d)
        {
            unsigned int ai = offset*(2*thid+1)-1;
            unsigned int bi = offset*(2*thid+2)-1;
            s_idata[bi] += s_idata[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { s_idata[n - 1] = 0; } // clear the last element
    for (unsigned int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            unsigned int ai = offset*(2*thid+1)-1;
            unsigned int bi = offset*(2*thid+2)-1;
            unsigned int t = s_idata[ai];
            s_idata[ai] = s_idata[bi];
            s_idata[bi] += t;
        }
    }
}

template <unsigned int blockSize>
__global__ void partition(
        unsigned int * g_offsetLessEquals,
        unsigned int * g_offsetGreater,
        float * g_idata,
        float * g_odata,
        float pivot,
        unsigned int nLeft,
        unsigned int n) {
    __shared__ unsigned int s_lessEquals[blockSize * 2];
    __shared__ unsigned int s_greater[blockSize * 2];

    __shared__ unsigned int s_offsetLessEquals;
    __shared__ unsigned int s_offsetGreater;

    unsigned int tid = threadIdx.x;

    unsigned int i = blockIdx.x * blockSize * 2 + 2 * tid;
    unsigned int j = blockIdx.x * blockSize * 2 + 2 * tid + 1;
    //unsigned int gridSize = blockSize*2*gridDim.x;

    bool f1, f2;
    if (i < n) {
        f1 = g_idata[i] <= pivot;
        f2 = not f1;
        // potential to avoid bank conflicts here
        s_lessEquals[2*tid] = f1;
        s_greater[2*tid] = f2;
    }
    else {
        f1 = false;
        f2 = false;
        s_lessEquals[2*tid] = 0;
        s_greater[2*tid] = 0;
    }

    bool f3, f4;
    if (j < n) {
        f3 = g_idata[j] <= pivot;
        f4 = not f3;
        // potential to avoid bank conflicts here
        s_lessEquals[2*tid+1] = f3;
        s_greater[2*tid+1] = f4;
    }
    else {
        f3 = false;
        f4 = false;
        s_lessEquals[2*tid+1] = 0;
        s_greater[2*tid+1] = 0;
    }

    __syncthreads();

    scan(s_lessEquals, tid, blockSize * 2 );
    scan(s_greater, tid, blockSize * 2);

    __syncthreads();

    // Avoid another kernel
    if (tid == blockSize - 1) {
        // result shared among kernel
        // atomicAdd returns old
        // exclusive scan does not include the last element
        s_offsetLessEquals = atomicAdd(g_offsetLessEquals, s_lessEquals[blockSize * 2 - 1] + f3);
        s_offsetGreater = atomicAdd(g_offsetGreater, s_greater[blockSize * 2 - 1] + f4);
    }

    __syncthreads();

    // avoiding warp divergence
    unsigned int indexA = (s_lessEquals[2*tid] + s_offsetLessEquals) * f1 +
                          (s_greater[2*tid] + s_offsetGreater + nLeft) * f2;

    unsigned int indexB = (s_lessEquals[2*tid+1] + s_offsetLessEquals) * f3 +
                          (s_greater[2*tid+1] + s_offsetGreater + nLeft) * f4;

    if (i < n) {
        g_odata[indexA] = g_idata[i];
    }

    if (j < n) {
        g_odata[indexB] = g_idata[j];
    }
}

int main(int argc, char** argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    unsigned int n = 1 << 13;
    unsigned int nLeft = 0;
    float cut = 0.5;

    float * h_idata = (float*)malloc(n * sizeof (float));
    float * h_odata = (float*)malloc(n * sizeof (float));

    srand(0);

    for (int i = 0; i < n; i++) {
        h_idata[i] = (float)(rand())/(float)(RAND_MAX);
        nLeft += h_idata[i] < cut;

        //printf("%f\n", h_idata[i]);
    }

    int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0));

    float * d_idata;
    float * d_odata;

    CUDA_CHECK(hipMalloc, (&d_idata, sizeof (float) * n));
    CUDA_CHECK(hipMalloc, (&d_odata, sizeof (float) * n));
    hipMemcpy(d_idata, h_idata, sizeof (float ) * n, hipMemcpyHostToDevice);

    unsigned int * d_offsetLessEquals;
    unsigned int * d_offsetGreater;

    CUDA_CHECK(hipMalloc,(&d_offsetLessEquals, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc,(&d_offsetGreater, sizeof(unsigned int)));

    CUDA_CHECK(hipMemset,(d_offsetLessEquals, 0, sizeof (unsigned int)));
    CUDA_CHECK(hipMemset,(d_offsetGreater, 0, sizeof (unsigned int)));
    CUDA_CHECK(hipMemset,(d_odata, 255, sizeof (float ) * n));

    hipEventRecord(start);

    partition<N_THREADS><<<
            nBlocks,
            N_THREADS,
            (N_THREADS + 2) * sizeof (unsigned int) * 4 + sizeof (unsigned int) * 2
            >>>(
            d_offsetLessEquals,
            d_offsetGreater,
            d_idata,
            d_odata,
            cut,
            nLeft,
            n);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_odata, d_odata, sizeof (float ) * n, hipMemcpyDeviceToHost);

    int sum = 0;

    /*for (int i = 0; i < n; ++i) {
        printf("%f ", h_idata[i]);
    }*/
    for (int i = 0; i < n; ++i) {
        printf("%f \n", h_odata[i]);
        //printf("%f ", h_idata[i]);
        /*if (h_odata[i] > cut) {
            throw std::runtime_error("Partition failed");
        }*/
    }

    printf("\n");

    hipFree(d_idata);
    hipFree(d_odata);
    hipFree(d_offsetGreater);
    hipFree(d_offsetLessEquals);

    free(h_idata);
    free(h_odata);

    hipDeviceReset();
}