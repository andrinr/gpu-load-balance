#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include "../../src/utils/condReduce.cuh"
#include "../../src/constants.h"
#include <thrust/partition.h>
// https://www.cse.chalmers.se/~tsigas/papers/GPU-Quicksort-jea.pdf
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/2_Concepts_and_Techniques/scan
// https://onlinelibrary.wiley.com/doi/epdf/10.1002/cpe.3611

#define N_THREADS 256
#define SHARED_MEMORY_BANKS 8
#define LOG_MEM_BANKS 3
#define CONFLICT_FREE_OFFSET(n) ((n) >> SHARED_MEMORY_BANKS + (n) >> (2 * LOG_MEM_BANKS))
// 2 data elements per thread
template <unsigned int blockSize>
__device__ void scan(volatile unsigned int * s_idata, unsigned int thid) {

    int ai = thid;
    int bi = thid + (blockSize);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = g_idata[ai];
    temp[bi + bankOffsetB] = g_idata[bi];

    int offset = 1;
    for (int d = blockSize>>1; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            s_idata[bi] += s_idata[ai];
        }
        offset *= 2;
    }
    if (thid==0) {
        s_idata[blockSize - 1 + CONFLICT_FREE_OFFSET(blockSize * 2 - 1)] = 0;
    }

    for (int d = 1; d < blockSize; d *= 2) {// traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            float t = s_idata[ai];
            s_idata[ai] = s_idata[bi];
            s_idata[bi] += t;
        }
    }  __syncthreads();
}

template <unsigned int blockSize>
__global__ void partition(
        unsigned int * g_totalLeft,
        unsigned int * g_totalRight,
        float * g_idata,
        float * g_odata,
        float pivot,
        unsigned int nLeft) {
    extern __shared__ unsigned int s_lqPivot[];
    extern __shared__ unsigned int s_gPivot[];
    extern __shared__ float s_res[];

    extern __shared__ unsigned int offsetLeq;
    extern __shared__ unsigned int offsetG;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize * 2)+threadIdx.x;
    //unsigned int gridSize = blockSize*2*gridDim.x;

    unsigned int f1 = g_idata[2*i] < pivot;
    s_lqPivot[2 * tid] = f1;
    s_gPivot[2 * tid] = 1-f1;

    unsigned int f2 = g_idata[2 * i + 1] < pivot;
    s_lqPivot[2 * tid + 1] = f2;
    s_gPivot[2 * tid + 1] = 1-f2;

    __syncthreads();

    scan<blockSize>(s_lqPivot, tid);
    scan<blockSize>(s_gPivot, tid);

    __syncthreads();

    // Avoid another kernel
    if (tid == 0) {
        offsetLeq = atomicAdd(g_totalLeft, s_lqPivot[blockSize * 2 - 1]);
        offsetG = atomicAdd(g_totalRight, s_gPivot[blockSize * 2 - 1]);
    }

    __syncthreads();

    // avoiding branch divergence
    g_odata[
            (s_lqPivot[2*tid] + offsetLeq) * f1 +
            (s_gPivot[2*tid] + offsetG + nLeft) * (1-f1)] = g_idata[2*i];

    g_odata[
            (s_lqPivot[2*tid+1] + offsetLeq) * f2 +
            (s_gPivot[2*tid+1] + offsetG + nLeft) * (1-f2)] = g_idata[2*i+1];

}


int main(int argc, char** argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 1 << 13;
    unsigned int nLeft = 0;
    float cut = 0.5;

    float * pos = (float*)malloc(n);
    for (int i = 0; i < n; i++) {
        pos[i] = (float)(rand())/(float)(RAND_MAX);
        nLeft += pos[i] < cut;
    }

    int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0));

    float * d_idata;
    float * d_odata;
    CUDA_CHECK(hipMalloc, (&d_idata, sizeof (float) * n));
    CUDA_CHECK(hipMalloc, (&d_odata, sizeof (float) * n));
    hipMemcpy(d_idata, pos, sizeof (float ) * n, hipMemcpyHostToDevice);

    unsigned int * d_totalLeq;
    unsigned int * d_totalG;

    hipMalloc(&d_totalLeq, sizeof(uint));
    hipMalloc(&d_totalG, sizeof(uint));

    hipMemset(&d_totalLeq, 0, sizeof(uint));
    hipMemset(&d_totalG, 0, sizeof(uint));

    hipEventRecord(start);

    partition<N_THREADS><<<
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (uint) * 2 >>>(
            d_totalLeq,
            d_totalG,
            d_idata,
            d_odata,
            cut,
            nLeft);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    hipMemcpy(pos, d_odata, sizeof (float ) * n, hipMemcpyDeviceToHost);

    int sum = 0;

    for (int i = 0; i < nLeft; ++i) {
        if (pos[i] > cut) {
            throw std::runtime_error("Error");
        }
    }

    std::cout << "is " << sum << " should be " << n / 2 << " \n";

    hipFree(d_idata);
    hipFree(d_odata);

    free(pos);

    hipDeviceReset();
}