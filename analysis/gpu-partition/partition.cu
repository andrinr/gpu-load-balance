#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include "../../src/utils/condReduce.cuh"
#include "../../src/constants.h"
#include <thrust/partition.h>
// https://www.cse.chalmers.se/~tsigas/papers/GPU-Quicksort-jea.pdf
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/2_Concepts_and_Techniques/scan
// https://onlinelibrary.wiley.com/doi/epdf/10.1002/cpe.3611


// 2 data elements per thread
// Code taken from: https://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
__device__ void scan(volatile unsigned int * s_idata, unsigned int thid, unsigned int n) {

    unsigned int offset = 1;
    for (unsigned int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d)
        {
            unsigned int ai = offset*(2*thid+1)-1;
            unsigned int bi = offset*(2*thid+2)-1;
            s_idata[bi] += s_idata[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { s_idata[n - 1] = 0; } // clear the last element
    for (unsigned int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            unsigned int ai = offset*(2*thid+1)-1;
            unsigned int bi = offset*(2*thid+2)-1;
            unsigned int t = s_idata[ai];
            s_idata[ai] = s_idata[bi];
            s_idata[bi] += t;
        }
    }

}

template <unsigned int blockSize>
__global__ void partition(
        unsigned int * g_offsetLessEquals,
        unsigned int * g_offsetGreater,
        float * g_idata,
        float * g_odata,
        float pivot,
        unsigned int nLeft) {
    __shared__ unsigned int s_lessEquals[blockSize * 2];
    __shared__ unsigned int s_greater[blockSize * 2];

    __shared__ unsigned int s_offsetLessEquals;
    __shared__ unsigned int s_offsetGreater;

    unsigned int tid = threadIdx.x;
    unsigned int n = blockSize * 2;
    unsigned int i = blockIdx.x * n + 2 * tid;
    unsigned int j = blockIdx.x * n + 2 * tid + 1;
    //unsigned int gridSize = blockSize*2*gridDim.x;

    bool f1 = g_idata[i] <= pivot;
    // potential to avoid bank conflicts here
    s_lessEquals[2*tid] = f1;
    s_greater[2*tid] = not f1;

    bool f2 = g_idata[j] <= pivot;
    s_lessEquals[2*tid+1] = f2;
    s_greater[2*tid+1] = not f2;

    __syncthreads();

    scan(s_lessEquals, tid, n);
    scan(s_greater, tid, n);

    __syncthreads();

    // Avoid another kernel
    if (tid == 0) {
        // result shared among kernel
        // atomicAdd returns old
        s_offsetLessEquals = atomicAdd(g_offsetLessEquals, s_lessEquals[n - 1]);
        s_offsetGreater = atomicAdd(g_offsetGreater, s_greater[n - 1]);
    }

    __syncthreads();

    // avoiding branch divergence
    unsigned int indexA = (s_lessEquals[2*tid] + s_offsetLessEquals) * f1 +
                          (s_greater[2*tid] + s_offsetGreater + nLeft) * (not f1);

    unsigned int indexB = (s_lessEquals[2*tid+1] + s_offsetLessEquals) * f2 +
                          (s_greater[2*tid+1] + s_offsetGreater + nLeft) * (not f2);

    g_odata[i] = (float) s_greater[2*tid];
    g_odata[j] = (float) s_greater[2*tid+1];


    //g_odata[indexA] = g_idata[i];
    //g_odata[indexB] = g_idata[j];

}

int main(int argc, char** argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 1 << 9;
    unsigned int nLeft = 0;
    float cut = 0.5;

    float * h_idata = (float*)malloc(n * sizeof (float));
    float * h_odata = (float*)malloc(n * sizeof (float));

    for (int i = 0; i < n; i++) {
        h_idata[i] = (float)(rand())/(float)(RAND_MAX);
        nLeft += h_idata[i] < cut;
    }

    int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0));
    printf("nBlocks: %d\n", nBlocks);

    float * d_idata;
    float * d_odata;

    printf("left %i \n", nLeft);
    CUDA_CHECK(hipMalloc, (&d_idata, sizeof (float) * n));
    CUDA_CHECK(hipMalloc, (&d_odata, sizeof (float) * n));
    hipMemcpy(d_idata, h_idata, sizeof (float ) * n, hipMemcpyHostToDevice);

    unsigned int * d_offsetLessEquals;
    unsigned int * d_offsetGreater;

    CUDA_CHECK(hipMalloc,(&d_offsetLessEquals, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc,(&d_offsetGreater, sizeof(unsigned int)));

    CUDA_CHECK(hipMemset,(d_offsetLessEquals, 0, sizeof (unsigned int)));
    CUDA_CHECK(hipMemset,(d_offsetGreater, 0, sizeof (unsigned int)));

    hipEventRecord(start);

    partition<N_THREADS><<<
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (unsigned int) * 8 + sizeof (unsigned int) * 2
            >>>(
            d_offsetLessEquals,
            d_offsetGreater,
            d_idata,
            d_odata,
            cut,
            nLeft);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    hipMemcpy(h_odata, d_odata, sizeof (float ) * n, hipMemcpyDeviceToHost);

    int sum = 0;

    /*for (int i = 0; i < n; ++i) {
        printf("%f ", h_idata[i]);
    }*/
    for (int i = 0; i < n; ++i) {
        printf("%f ", h_odata[i]);
        printf("%f ", h_idata[i]);
        /*if (h_odata[i] > cut) {
            throw std::runtime_error("Partition failed");
        }*/
    }

    printf("\n");

    hipFree(d_idata);
    hipFree(d_odata);
    hipFree(d_offsetGreater);
    hipFree(d_offsetLessEquals);

    free(h_idata);
    free(h_odata);

    hipDeviceReset();
}