#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include "../../src/utils/condReduce.cuh"
#include "../../src/constants.h"
#include <thrust/partition.h>
// https://www.cse.chalmers.se/~tsigas/papers/GPU-Quicksort-jea.pdf
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/2_Concepts_and_Techniques/scan
// https://onlinelibrary.wiley.com/doi/epdf/10.1002/cpe.3611


// 2 data elements per thread
// Code taken from: https://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
__device__ void scan(volatile unsigned int * s_idata, unsigned int thid, unsigned int n) {

    int offset = 1;
    for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            s_idata[bi] += s_idata[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { s_idata[n - 1] = 0; } // clear the last element
    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            float t = s_idata[ai];
            s_idata[ai] = s_idata[bi];
            s_idata[bi] += t;
        }
    }
}

template <unsigned int blockSize>
__global__ void partition(
        unsigned int * g_offsetLessEquals,
        unsigned int * g_offsetGreater,
        float * g_idata,
        float * g_odata,
        float pivot,
        unsigned int nLeft) {
    extern __shared__ unsigned int s_lessEquals[];
    extern __shared__ unsigned int s_greater[];

    __shared__ unsigned int s_offsetLessEquals;
    __shared__ unsigned int s_offsetGreater;

    unsigned int tid = threadIdx.x;
    unsigned int n = blockSize * 2;
    unsigned int i = blockIdx.x*n+threadIdx.x;
    unsigned int j = blockIdx.x*n+threadIdx.x + blockSize;
    //unsigned int gridSize = blockSize*2*gridDim.x;

    unsigned int f1 = g_idata[i] < pivot;
    // potential to avoid bank conflicts here
    s_lessEquals[2*tid] = f1;
    s_greater[2*tid] = 1-f1;

    unsigned int f2 = g_idata[j] < pivot;
    s_lessEquals[2*tid+1] = f2;
    s_greater[2*tid+1] = 1-f2;

    __syncthreads();

    scan(s_lessEquals, tid, n);
    scan(s_greater, tid, n);

    __syncthreads();

    // Avoid another kernel
    if (tid == 0) {
        // result shared among kernel
        s_offsetLessEquals = atomicAdd(g_offsetLessEquals, s_lessEquals[n - 1]);
        s_offsetGreater = atomicAdd(g_offsetGreater, s_greater[n - 1]);
    }

    __syncthreads();

    // avoiding branch divergence
    unsigned int indexA = (s_lessEquals[2*tid] + s_offsetLessEquals) * f1 +
                          (s_greater[2*tid] + s_offsetGreater + nLeft) * (1-f1);
    unsigned int indexB = (s_lessEquals[2*tid+1] + s_offsetLessEquals) * f2 +
                          (s_greater[2*tid+1] + s_offsetGreater + nLeft) * (1-f2);

    //g_odata[indexA] = g_idata[i];
    //g_odata[indexB] = g_idata[j];

    g_odata[i] = (float) s_lessEquals[2*tid];
    g_odata[j] = (float) s_lessEquals[2*tid+1];
}

int main(int argc, char** argv) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 1 << 9;
    unsigned int nLeft = 0;
    float cut = 0.5;

    float * h_idata = (float*)malloc(n * sizeof (float));
    float * h_odata = (float*)malloc(n * sizeof (float));

    for (int i = 0; i < n; i++) {
        h_idata[i] = (float)(rand())/(float)(RAND_MAX);
        nLeft += h_idata[i] < cut;
    }

    int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0));
    printf("nBlocks: %d\n", nBlocks);

    float * d_idata;
    float * d_odata;

    printf("left %i \n", nLeft);
    CUDA_CHECK(hipMalloc, (&d_idata, sizeof (float) * n));
    CUDA_CHECK(hipMalloc, (&d_odata, sizeof (float) * n));
    hipMemcpy(d_idata, h_idata, sizeof (float ) * n, hipMemcpyHostToDevice);

    unsigned int * d_offsetLessEquals;
    unsigned int * d_offsetGreater;

    CUDA_CHECK(hipMalloc,(&d_offsetLessEquals, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc,(&d_offsetGreater, sizeof(unsigned int)));

    CUDA_CHECK(hipMemset,(d_offsetLessEquals, 0, sizeof (unsigned int)));
    CUDA_CHECK(hipMemset,(d_offsetGreater, 0, sizeof (unsigned int)));

    hipEventRecord(start);

    partition<N_THREADS><<<
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (unsigned int) * 8 + sizeof (unsigned int) * 2
            >>>(
            d_offsetLessEquals,
            d_offsetGreater,
            d_idata,
            d_odata,
            cut,
            nLeft);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    hipMemcpy(h_odata, d_odata, sizeof (float ) * n, hipMemcpyDeviceToHost);

    int sum = 0;

    for (int i = 0; i < n; ++i) {
        printf("%f ", h_odata[i]);
        /*if (h_odata[i] > cut) {
            throw std::runtime_error("Partition failed");
        }*/
    }

    printf("\n");

    hipFree(d_idata);
    hipFree(d_odata);
    hipFree(d_offsetGreater);
    hipFree(d_offsetLessEquals);

    free(h_idata);
    free(h_odata);

    hipDeviceReset();
}