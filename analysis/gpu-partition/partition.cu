#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include "../../src/utils/condReduce.cuh"

// https://www.cse.chalmers.se/~tsigas/papers/GPU-Quicksort-jea.pdf
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/2_Concepts_and_Techniques/scan
// https://onlinelibrary.wiley.com/doi/epdf/10.1002/cpe.3611

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

// 2 data elements per thread
template <unsigned int blockSize>
__device__ void d_scan(volatile uint * s_idata, uint thid, int n) {

    for (int d = n>>1; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            s_idata[bi] += s_idata[ai];
        }
        offset *= 2;
    }
    if (thid==0) {
        s_idata[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    }

    for (int d = 1; d < n; d *= 2) {// traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            float t = s_idata[ai];
            s_idata[ai] = s_idata[bi];
            s_idata[bi] += t;
        }
    }  __syncthreads();
}


template <unsigned int blockSize>
__global__ void partition(int totalLeft, int totalRight, uint *  offsetLeq, uint *  offsetG, float * g_idata, float * g_odata, float pivot) {
    extern __shared__ uint s_lqPivot[];
    extern __shared__ uint s_gPivot[];
    extern __shared__ float s_res[];

    extern __shared__ uint offsetLeq;
    extern __shared__ uint offsetG;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize * 2)+threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    uint l_offsetLeq = offsetLeq[blockIdx.x];
    uint l_offsetG = offsetG[blockIdx.x];

    // Avoid another kernel
    if (tid == 0) {
        offsetLeq = atomicAdd(&totalLeft, l_offsetLeq);
        offsetG = atomicAdd(&totalRight, l_offsetG);
    }

    uint f1 = g_idata[2*i] < pivot
    s_lqPivot[2 * tid] = f;
    s_gPivot[2 * tid] = 1-f;

    uint f2 = g_idata[2 * i + 1] < pivot
    s_lqPivot[2 * tid + 1] = f;
    s_gPivot[2 * tid + 1] = 1-f;

    __syncthreads();

    uint offLq = scan(s_lqPivot, tid);
    uint offG = scan(s_gPivot, tid);

    __syncthreads();

    // avoiding branch divergence
    g_odata[
            (s_lqPivot[2*tid] + offsetLeq) * f1 +
            (s_gPivot[2*tid] + offsetG) * (1-f1)] = g_idata[2*i];

    g_odata[
            (s_lqPivot[2*tid+1] + offsetLeq) * f2 +
            (s_gPivot[2*tid+1] + offsetG) * (1-f2)] = g_idata[2*i+1];

}



template <unsigned int blockSize>
extern __device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize, bool leq>
extern __global__ void reduce(float *g_idata, uint *g_odata, float cut, int n) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize) + threadIdx.x;
    unsigned int gridSize = blockSize*gridDim.x;
    sdata[tid] = 0;

    if (leq){
        sdata[2*tid] += (g_idata[2*i] <= cut);
        sdata[2*tid+1] += (g_idata[2*i+1] <= cut);
    } else {
        sdata[2*tid] += (g_idata[2*i] > cut);
        sdata[2*tid+1] += (g_idata[2*i+1] > cut);
    }

    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        } __syncthreads();
    }
    if (tid < 32) {
        warpReduce<blockSize>(sdata, tid);
    }
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}



int main(int argc, char** argv) {

    const int N_THREADS = 512;
    const int ELEMENTS_PER_THREAD = 32;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int n = 1 << 13;
    float cut = 0.5;

    float * pos = (float*)malloc(n);
    for (int i = 0; i < n; i++) {
        pos[i] = i;
    }

    // Can increase speed by another factor of around two
    int elementsPerThread = 32;
    int nBlocks = (int) ceil((float) n / (N_THREADS * 2.0));
    printf("nThreads %i, nBlocks %i, n %i \n", nThreads, nBlocks, n);

    float * d_idata;
    float * d_odata;
    hipMalloc(&d_idata, sizeof (float) * n);
    hipMalloc(&d_odata, sizeof (float) * n);
    hipMemcpy(d_idata, pos, sizeof (float ) * n, hipMemcpyHostToDevice);

    uint * countA = (uint*)malloc(nBlocks * sizeof(uint));
    uint * countB = (uint*)malloc(nBlocks * sizeof(uint));

    hipEventRecord(start);

    reduce<N_THREADS, true>(
            g_idata,
            countA,
            cut,
            n,
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (uint) * 2
    );

    reduce<N_THREADS, false>(
            g_idata,
            countB,
            cut,
            n,
            nBlocks,
            N_THREADS,
            N_THREADS * sizeof (uint) * 2
    );

    partition<nThreads><<<
            nBlocks,
            nThreads,
            nThreads * sizeof (uint) * 2 >>>(
            countA,
            countB,
            g_idata,
            g_odata,
            cut;

    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    hipMemcpy(h_sums, d_sums, sizeof (int ) * nBlocks, hipMemcpyDeviceToHost);

    int sum = 0;

    for (int i = 0; i < nBlocks; ++i) {
        printf("sum %i \n", sum);
        sum += h_sums[i];
    }

    std::cout << "is " << sum << " should be " << n / 2 << " \n";

    hipFree(d_particles);
    hipFree(d_sums);

    free(h_sums);
    free(pos);

    hipDeviceReset();
}