
#include <hip/hip_runtime.h>
#include "math.h"
#include "stdio.h"
#include <experimental/random>
#include <iostream>
#include <fstream>

__global__ void split(int N, int * pos){
	int stride = blockDim.x;
	int index = threadIdx.x;
}

int main() 
{
	int N = 10<<16; // 60k  Elemets
	int *xpos;
	int *domain;

	hipMallocManaged(&xpos, N*sizeof(double));
	hipMallocManaged(&domain, N*sizeof(int));
	
	int pos [N*3];
	int vel [N*3];

	for (int i = 0; i < N; i++){
		// xpos
		pos[3*i] = std::experimental::randint(INT_MIN, INT_MAX);
		xpos[i] = pos[3*i];
		domain[0] = 0;
		// ypos
		pos[3*i + 1] = std::experimental::randint(INT_MIN, INT_MAX);
		// zpos
		pos[3*i + 2] = std::experimental::randint(INT_MIN, INT_MAX);

		// xvel
		vel[3*i] = 0;
		// yvel
		vel[3*i+1] = 0;
		// zpos
		vel[3*i+2] = 0;
	}
	
	
	split<<<1,256>>>(N, xpos); 

	hipDeviceSynchronize();
	
	printf("calculated one step");

	hipFree(xpos);
	
	remove( "out.dat" );
	std::ofstream Data("out.dat");
	
	for (int i = 0; i < N; i++){
		Data << pos[3*i] << " " << pos[3*i +1] << " " << pos[3*i+2] << "\n";
	}

	Data.close();
    	return 0;
}

